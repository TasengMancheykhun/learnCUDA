// nvcc -std=c++11 -o exe 29_sum_array.cu




#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>


__global__ void sumArraysOnGPU(int *A, int *B, int *C, const int N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N)
        C[tid] = A[tid] + B[tid];
}


void sumArraysOnHost(int *A, int *B, int *C, const int N){
    for (int idx=0; idx<N; idx++)
        C[idx] = A[idx] + B[idx];    
}


int main(){
   int N=1<<25;
//   int N=1000;
   
   printf("Vector size %d\n", N);

   size_t nBytes = N*sizeof(int);
   printf("nBytes = %zu \n", nBytes);

   int *h_A, *h_B, *h_C, *h_device_result;
   int *d_A, *d_B, *d_C;

   h_A = (int *) malloc(nBytes);
   h_B = (int *) malloc(nBytes);
   h_C = (int *) malloc(nBytes);

   h_device_result = (int *) malloc(nBytes);

   hipMalloc((int **) &d_A, nBytes);
   hipMalloc((int **) &d_B, nBytes);
   hipMalloc((int **) &d_C, nBytes);

   for (int i=0; i<N; i++){
      h_A[i] = i+1;
      h_B[i] = i+1;
      h_C[i] = 0;
      h_device_result[i]=0;
   }

    // CPU addition

    auto start_time = std::chrono::high_resolution_clock::now();
    sumArraysOnHost(h_A, h_B, h_C, N);
    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration_ns = std::chrono::duration_cast<std::chrono::nanoseconds > (end_time - start_time).count();
    
    double seconds_cpu = duration_ns/1000000000.0;
    
    std::cout<<"Time taken by cpu : "<< seconds_cpu <<" seconds"<<std::endl;
    std::cout<<"Time taken by cpu : "<< duration_ns <<" nanoseconds"<<std::endl;
    
    int time_host = (int) duration_ns; 
    
    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, nBytes, hipMemcpyHostToDevice);
    
    // GPU addition
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock -1)/threadsPerBlock;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    sumArraysOnGPU<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
   
    hipEventSynchronize(stop);
 
    float milliseconds_device;

    hipEventElapsedTime(&milliseconds_device, start, stop);


    long long nanoseconds_device = static_cast<long long>(milliseconds_device * 1e6);

    printf("\nTime taken by device (GPU): %lld nanoseconds\n", nanoseconds_device);
    
//  printf("Time taken by device(gpu):%.0f seconds\n",milliseconds_device/1000);
    
    //copy kernel result back to host side
    hipMemcpy(h_device_result, d_C, nBytes,hipMemcpyDeviceToHost);
 
    int time_device = (int) nanoseconds_device;    
   

    int diff = time_host-time_device;
 
    printf("Difference: %d nanoseconds \n",diff);

    float percent_gain=(float(diff)/float(time_host))*100;

    printf("Percent gain: %f % \n",percent_gain);


//    printf("Printing Result\n");    
//    for(int i=0;i<5;i++){
//    printf("i=%d,sum=%d\n",i,h_device_result[i]);
//    }
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
    
}



