#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void atomicKernel(int* shared_counter)
{
  // Each thread tries tp increment the shared shared_counter
  atomicAdd(shared_counter,1);
  *shared_counter++;
}

int main()
{
    int h_counter = 0;
    int* d_counter;

    hipMalloc((void**)&d_counter, sizeof(int));
    hipMemcpy(d_counter, &h_counter, sizeof(int), hipMemcpyHostToDevice);

    atomicKernel<<<1, 1024>>> (d_counter);

    hipDeviceSynchronize();

    hipMemcpy(&h_counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_counter);

    printf("Final counter value: %d\n", h_counter);

    return 0;
}
