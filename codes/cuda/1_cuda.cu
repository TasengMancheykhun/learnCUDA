#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_arrays(int *c, const int *a, const int *b, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<size){                                                                        // for handling the error of more threads than array elements
        c[i] = a[i] + b[i];
        printf("This thread has done processing... Thread is %d \n",i);
    }
    else{
         printf("This thread has NOT done processing... Thread is %d \n",i);
    }
}


int main()
{
   const int size = 5;
//   int a[size] = {1,2,3,4,5};
//   int b[size] = {1,2,3,4,5};

   int a[size], b[size];
   printf("Enter 5 elements for array 'a': \n");
   for(int i = 0; i < size; i++){
       scanf("%d",&a[i]);
   }

   printf("Enter 5 elements for array 'b': \n");
   for(int i = 0; i < size; i++){
       scanf("%d",&b[i]);
   }


   //Allocate memory on the device for array C
   int *d_c;
   hipMalloc((void **)&d_c, size * sizeof(int));

   //Allocate memory on the device for array A and B
   int *d_a, *d_b;
   hipMalloc((void **)&d_a, size * sizeof(int));
   hipMalloc((void **)&d_b, size * sizeof(int));

   hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

   // Launch the kernel with one block and size threads
//   add_arrays<<<1,5>>>(d_c, d_a, d_b, size);    // thread equal to number of array elements
//   add_arrays<<<1,3>>>(d_c, d_a, d_b, size);    // thread number less than number of array elements, no error
   add_arrays<<<1,8>>>(d_c, d_a, d_b, size);      // thread number larger than number of array elements, gives error


   // Add synchronization to ensure kernel execution completes
   hipDeviceSynchronize();

   //copy the result back from the device to the host
   int *c = (int*)malloc(size * sizeof(int));

   hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

   //Print the result;
   for(int i = 0; i < size; i++)
   {
       printf("%d + %d = %d \n", a[i], b[i], c[i]);
   }

   //Free the memory on device
   hipFree(c);
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

   return 0;
}


