/*
3) Find the minimum/maximum in an array without allowing for race conditions to occur:
   atomicMax(max_val, arr[idx]);
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024

__global__ void atomicKernel(int *shared_counter_max, int *shared_counter_min, int *d_array)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    if (idx < N)
        {
            atomicMax(shared_counter_max, d_array[idx]);
            
            atomicMin(shared_counter_min, d_array[idx]);            

            shared_counter_max++;
            shared_counter_min++;

        }
}


int main()
{
    int h_array[N];
    int *d_array;
    int *d_result_max;
    int *d_result_min;
    int h_result_max;
    int h_result_min;
    

    for (int i=0; i<N; i++)
    {
        h_array[i]=i+10;
    }

    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, h_array, N*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_result_max, sizeof(int));
    hipMalloc(&d_result_min, sizeof(int));

    atomicKernel<<<1, 1024>>> (d_result_max, d_result_min, d_array);

    hipDeviceSynchronize();

    hipMemcpy(&h_result_max, d_result_max, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_result_min, d_result_min, sizeof(int), hipMemcpyDeviceToHost);

    printf("Max value: %d\n", h_result_max);
    printf("Min value: %d\n", h_result_min);

    hipFree(d_array);
    hipFree(d_result_max);
    hipFree(d_result_min);

    return 0;
}
