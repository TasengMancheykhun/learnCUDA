/*
Modify the code to use the formula:
result = (n*n+1)/2

Note: Calculation should still happen in the kernel function
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024

__global__ void atomicKernel(int *shared_counter, int *d_array)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (idx < N)
      {
      atomicAdd(shared_counter, d_array[idx]);
      }
//  *shared_counter++;
}



int main()
{
    int h_array[N];
    int *d_array;
    int *d_result;
    int h_result;

    for (int i=0;i<N;i++)
    {
      h_array[i]=i+1;
    }

    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, h_array, N*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_result, sizeof(int));

    atomicKernel<<<1, 1024>>> (d_result, d_array);

    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Final value: %d\n", h_result);

    hipFree(d_array);
    hipFree(d_result);

    return 0;
}
