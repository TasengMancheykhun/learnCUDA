/* 
Create an array containing 1024 elements
Initialize it with values 1 to 1024 on the host
In the kernel code, add values of all the array elements to calculate the sum of all the array elements

result = array[0] + array[1] + array[2] + ... array[1023]

In the host code, display the result
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024

__global__ void atomicKernel(int *shared_counter, int *d_array)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (idx < N)
      {
      atomicAdd(shared_counter, d_array[idx]);
      }
//  *shared_counter++;
}



int main()
{
    int h_array[N];
    int *d_array;
    int *d_result;
    int h_result;

    for (int i=0;i<N;i++)
    {
      h_array[i]=i+1;
    }

    hipMalloc(&d_array, N*sizeof(int));
    hipMemcpy(d_array, h_array, N*sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_result, sizeof(int));

    atomicKernel<<<1, 1024>>> (d_result, d_array);

    hipDeviceSynchronize();

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Final value: %d\n", h_result);

    hipFree(d_array);
    hipFree(d_result);

    return 0;
}
