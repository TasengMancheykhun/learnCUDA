#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel(void){
    printf("Hello from GPU 1 !... \n");
    printf("Hello from GPU 2 !... \n");
    printf("Hello from GPU 3 !... \n");
    printf("Hello from GPU 4 !... \n");
    printf("Hello from GPU 5 !... \n");
    printf("Hello from GPU 6 !... \n");
}


int main(){
   kernel<<<1,1>>> ();
   printf("Hello from CPU 1 !... \n");
   printf("Hello from CPU 2 !... \n");
   printf("Hello from CPU 3 !... \n");
   printf("Hello from CPU 4 !... \n");
   printf("Hello from CPU 5 !... \n");
   printf("Hello from CPU 6 !... \n");


   return 0;

}
