
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 100

__global__ void evenodd(int *d_arr, int *d_even, int *d_odd){
  
  __shared__ int shareddata[N/4];  

  int idx = blockIdx.x*blockDim.x+threadIdx.x;

  if(idx<N){
      shareddata[threadIdx.x] = d_arr[idx];
  }
  else{
     shareddata[threadIdx.x] = 0;
  }
  
  __syncthreads();

  if (threadIdx.x==0){
      int neven=0, nodd=0;
      for(int i=0; i<N/4; i++){
          if (shareddata[i]%2==0){
              neven++;
          }
          else{
              nodd++;
          }
      }
      d_even[blockIdx.x]=neven;
      d_odd[blockIdx.x]=nodd; 
  } 
  
}


int main(){

  int h_arr[N];
  for (int i=0; i<N; i++){
      h_arr[i] = i+1;  
  }

  int *d_arr, *d_even, *d_odd;
  
  hipMalloc(&d_arr, N*sizeof(int));
  hipMalloc(&d_even, N/25*sizeof(int));  
  hipMalloc(&d_odd, N/25*sizeof(int));  

  hipMemcpy(d_arr, h_arr, N*sizeof(int), hipMemcpyHostToDevice);
  
  int threadperblock = 25;
  int blockpergrid = (N + threadperblock - 1)/threadperblock;

  evenodd<<<blockpergrid, threadperblock>>>(d_arr, d_even, d_odd);
  hipDeviceSynchronize(); 
 

  int *h_even = (int *)malloc(N/25*sizeof(int));
  int *h_odd = (int *)malloc(N/25*sizeof(int));
  
  hipMemcpy(h_even, d_even, N/25*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(h_odd, d_odd, N/25*sizeof(int), hipMemcpyDeviceToHost);

   
  printf("Even \n");
  for(int i=0;i<N/25;i++){
      printf("%d ",h_even[i]);
  } 
  printf("\n");


  printf("Odd \n");
  for(int i=0;i<N/25;i++){
      printf("%d ",h_odd[i]);
  } 
  printf("\n");
 



 
  int heven=0, hodd=0;
  for(int i=0;i<N/25;i++){
      heven += h_even[i];
      hodd += h_odd[i];
  } 

  printf("No. of even is: %d\n", heven);
  printf("No. of odd is: %d\n", hodd);

  hipFree(d_arr);
  hipFree(d_even);
  hipFree(d_odd);

  free(h_even);
  free(h_odd);
  return 0;
}
