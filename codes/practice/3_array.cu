
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 5

__global__ void varadd(int *a, int *b, int *c)
{
   int idx = blockIdx.x*blockDim.x+threadIdx.x;

   if (idx < N)
       c[idx] = a[idx] + b[idx];   

}


int main()
{
  int h_a[N]={1,2,3,4,5};
  int h_b[N]={5,4,3,2,1};
  int h_c[N];

  int *d_a, *d_b, *d_c;

  hipMalloc(&d_a, N*sizeof(int));
  hipMalloc(&d_b, N*sizeof(int));
  hipMalloc(&d_c, N*sizeof(int));
 
  hipMemcpy(d_a, &h_a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, N*sizeof(int), hipMemcpyHostToDevice);

  varadd<<<1,5>>>(d_a, d_b, d_c);
    
  hipMemcpy(&h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);    

  for (int i=0;i<N;i++)
  {
    printf("%d ",h_c[i]);
  }
  printf("\n"); 
 
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
