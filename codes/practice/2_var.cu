
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void varadd(int *a, int *b, int *c)
{
   *c=*a+*b;   

}


int main()
{
  int h_a=6;
  int h_b=9;
  int h_c;
  int *d_a, *d_b, *d_c;

  hipMalloc(&d_a, sizeof(int));
  hipMalloc(&d_b, sizeof(int));
  hipMalloc(&d_c, sizeof(int));
 
  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

  varadd<<<1,1>>>(d_a, d_b, d_c);
    
  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);    

  printf("Sum: %d",h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
