#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<chrono>
#include<hip/hip_runtime.h>
#define N 32

__global__ void add(int *d_a,int *d_b,int *d_c){
  int idx = blockDim.x*blockIdx.x+threadIdx.x;

  if (idx<N){
    d_c[idx] = d_a[idx] + d_b[idx];
  }


}



int main(){

   int *d_a, *d_b, *d_c;

   hipMallocManaged(&d_a, N*sizeof(int));
   hipMallocManaged(&d_b, N*sizeof(int));
   hipMallocManaged(&d_c, N*sizeof(int));
   
   for(int i=0; i<N; i++){
     d_a[i] = i+1;
     d_b[i] = i+1;
   }
   
   threadsperblock = 8;
   blockspergrid = (N+threadsperblock-1)/threadsperblock;
   auto start = std::chrono::high_resolution_clock::now();
   add<<<blockspergrid,threadsperblock>>>(d_a, d_b, d_c);
   auto end = std::chrono::high_resolution_clock::now();

   auto duration = std::chrono::duration<double, std::milli>(end-start).count();

   printf("Result: \n");
   for(int i=0; i<N; i++){
       printf("%d ",d_c[i]);
   }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}

