
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 10

__global__ void addarray(int *d_a, int *d_b, int *d_c){

  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if (idx<N){
    d_c[idx] = d_a[idx] + d_b[idx];
  }
}


int main(){
  
  int *d_a, *d_b, *d_c;

  hipMallocManaged(&d_a, N*sizeof(int));
  hipMallocManaged(&d_b, N*sizeof(int));
  hipMallocManaged(&d_c, N*sizeof(int));
  
  for(int i=0;i<N;i++){
    d_a[i] = i+1;
    d_b[i] = i+2;
  } 

  printf("a: ");
  for(int i=0;i<N;i++){
    printf("%d ",d_a[i]);
  }
  printf("\n");

  printf("b: ");
  for(int i=0;i<N;i++){
    printf("%d ",d_b[i]);
  }
  printf("\n");


  int threadsperblock=32;
  int blockspergrid = (N+threadsperblock-1)/threadsperblock;
  addarray<<<blockspergrid,threadsperblock>>>(d_a, d_b, d_c);
  hipDeviceSynchronize();  

  for(int i=0;i<N;i++){
      printf("%d ",d_c[i]);
  }  
  printf("\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
