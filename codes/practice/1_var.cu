
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void addvar(int *a, int *b, int *c)
{
    *c = *a + *b;
}



int main()
{
  int h_a=5;
  int h_b=8;
  int h_c;

  int *d_a, *d_b, *d_c;

  hipMalloc(&d_a, sizeof(int));
  hipMalloc(&d_b, sizeof(int));
  hipMalloc(&d_c, sizeof(int));
  
  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

  addvar<<<1,1>>>(d_a, d_b, d_c);
  
  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("Ans: %d \n",h_c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}

/*

malloc ---

(int *) malloc(sizeof(int))

malloc dynamically allocates a block of memory with the specified size. It returns a pointer of type void which can be casted into a pointer of any form. This pointer points to the beginning of the block.   


cudaMalloc ---  

cudaMalloc(&d_a, sizeof(int))

cudaMalloc dynamically allocates a block of memory in the device memory. It returns a pointer that points to the allocated memory in the device.


cudaMemcpy ---

cudaMemcpy(d_a, &h_a, sizeof(int), cudaMemcpyHostToDevice)

cudaMemcpy copies sizeof(int) bytes of data from memory area pointed by pointer &h_a to memory area pointed by pointer d_b. 

*/

