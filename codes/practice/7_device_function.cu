
#include <hip/hip_runtime.h>
#include <stdio.h>


__device__ int reduce_sum(int marks[])
{
  int sum=0;

  for (int i=0; i<3; i++){
      sum += marks[i];
  }
  
  return sum;
}


__global__ void sum(int *d_marks, int *total){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx<5){
        int marks[3];
        for (int i=0;i<3;i++){
          marks[i] = d_marks[idx*3+i];
        }
        total[idx] = reduce_sum(marks);
    }
}


int main(){
  int marks[5][3] = {
                {55,60,70},
                {71,81,76},
                {74,65,64},
                {78,80,77},
                {72,75,67}
                };

  int *d_marks;
  int *total;
  int result[5];

  hipMalloc(&d_marks, 5*3*sizeof(int));
  hipMalloc(&total, 5*sizeof(int));
    
  hipMemcpy(d_marks, marks, 5*3*sizeof(int), hipMemcpyHostToDevice);

  sum<<<1,5>>>(d_marks, total);

  hipMemcpy(result, total, 5*sizeof(int), hipMemcpyDeviceToHost); 
  
  printf("Result: \n");
  for (int i=0; i<5; i++){
    printf("%d ",result[i]);
  }
  printf("\n");

  return 0;
}
