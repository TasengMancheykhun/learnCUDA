
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 16


__global__ void add(int *da, int *db, int *dc){
  
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  int stride = blockDim.x*gridDim.x;  

  if (idx<N){
     for (int i=idx; i<N; i+=stride){
     dc[i] = da[i]+db[i];
     }
  }
}


int main(){
  
  int a[N];
  int b[N];
  int c[N];

  srand((unsigned int) time (NULL));

  for (int i=0;i<N;i++){
    a[i] = rand()%N;
    b[i] = rand()%N;
  }

  printf("a: \n");
  for (int i=0; i<N; i++){
    printf("%d ",a[i]);
  } 
  printf("\n"); 

  printf("b: \n");
  for (int i=0; i<N; i++){
    printf("%d ",b[i]);
  }  
  printf("\n"); 

  
  int *da, *db, *dc;

  hipMalloc(&da,N*sizeof(int));
  hipMalloc(&db,N*sizeof(int));
  hipMalloc(&dc,N*sizeof(int));

  hipMemcpy(da,a,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(db,b,N*sizeof(int),hipMemcpyHostToDevice);

  int threadsperblock = 4; 
  int blockspergrid = (N+threadsperblock-1)/threadsperblock;

  add<<<threadsperblock, blockspergrid>>>(da, db, dc);   
  hipDeviceSynchronize();

  hipMemcpy(c,dc,N*sizeof(int),hipMemcpyDeviceToHost); 

  printf("c: \n");
  for (int i=0; i<N; i++){
    printf("%d ",c[i]);
  }  
  printf("\n"); 

  hipFree(da);
  hipFree(db);
  hipFree(dc);

  return 0;
}
