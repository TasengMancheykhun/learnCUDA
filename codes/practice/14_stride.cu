
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 16
   
__global__ void add(int *da, int *db, int *dc){
  
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  
  int stride = blockDim.x * gridDim.x;
  
  for(int i=idx; i<N; i+=stride){
    dc[i] = da[i] + db[i];
  }
 
}

int main(){

  int ha[N];
  int hb[N];
  int hc[N];

  int *da, *db, *dc;

  
  srand((unsigned int) time (NULL));

  for (int i=0; i<N; i++){
    ha[i] = rand()%N;
    hb[i] = rand()%N;
  }

  hipMalloc(&da, N*sizeof(int));
  hipMalloc(&db, N*sizeof(int));
  hipMalloc(&dc, N*sizeof(int));


  hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(db, hb, N*sizeof(int), hipMemcpyHostToDevice);

  int threadsperblock = 4;
  int blockspergrid = (N+threadsperblock-1)/threadsperblock;

  add<<<blockspergrid,threadsperblock>>>(da, db, dc);
  
  hipMemcpy(hc, dc, N*sizeof(int), hipMemcpyDeviceToHost);


  printf("a: \n");
  for (int i=0; i<N; i++){
    printf("%d ",ha[i]);
  } 
  printf("\n");


  printf("b: \n");
  for (int i=0; i<N; i++){
    printf("%d ",hb[i]);
  } 
  printf("\n");



  printf("Result: \n");
  for (int i=0; i<N; i++){
    printf("%d ",hc[i]);
  } 
  printf("\n");

 
  hipFree(da);
  hipFree(db);
  hipFree(dc);

  return 0;
}


