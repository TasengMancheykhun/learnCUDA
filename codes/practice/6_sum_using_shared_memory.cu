#include "hip/hip_runtime.h"
#include<stdio.h>
#define N 48

__global__ void sum(int *d_input, int *d_output){
  
  int idx = blockIdx.x*blockDim.x + threadIdx.x;

  __shared__ int shareddata[N/8];

  if (idx < N)
  {
     shareddata[threadIdx.x] = d_input[idx]; 
  }
  else
  {
     shareddata[threadIdx.x] = 0;
  }

  __syncthreads();

  if (threadIdx.x == 0)
  {
     int sum=0;
     for (int i=0;i<blockDim.x;i++)
     {
         sum+=shareddata[i];
     }
     d_output[blockIdx.x] = sum;
  }

}


int main()
{
  int h_input[N];
  
  for(int i=0; i<N; i++){
     h_input[i] = i+1;
  }  
  
  hipMalloc(&d_input, N*sizeof(int));
  hipMalloc(&d_output, N/6*sizeof(int));
  hipMemcpy(d_input, h_input, N*sizeof(int), hipMemcpyHostToDevice);

  int threadsperblock = 8;
  int blockspergrid = (N + threadsperblock - 1)/threadsperblock;
  
  sum<<<blockspergrid, threadsperblock>>>(d_input,d_output);
  
  hipMemcpy(h_output, d_output, N/6*sizeof(int), hipMemcpyDeviceToHost);
  
      
  
 
  return 0;
}
