#include<stdio.h>
#include<hip/hip_runtime.h>
#define N 1000

__global__ void minmax(int *d_A, int *d_result_min, int *d_result_max)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N)
    {
        atomicMin(d_result_min, d_A[idx]);
        atomicMax(d_result_max, d_A[idx]);
    }
}


int main(){
  int h_A[N], h_result_min = 0, h_result_max = 0;

  // Initialize array
  for (int i = 0; i < N; i++)
  {
    h_A[i] = i+1;     // Array values: 1,2,3,....N
  }

  int *d_A, *d_result_min, *d_result_max;

  hipMalloc(&d_A, N*sizeof(int));
  hipMalloc(&d_result_min, sizeof(int));  
  hipMalloc(&d_result_max, sizeof(int));

  hipMemcpy(d_A, h_A, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_result_min, &h_A[0], sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_result_max, &h_A[0], sizeof(int), hipMemcpyHostToDevice);

  int threadsperblock = 1024;
  int blockspergrid = (N + threadsperblock - 1)/threadsperblock;
  printf("t=%d, b=%d\n", threadsperblock, blockspergrid);

  minmax<<<blockspergrid, threadsperblock>>>(d_A, d_result_min, d_result_max);
  hipDeviceSynchronize();

  hipMemcpy(&h_result_min, d_result_min, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&h_result_max, d_result_max, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("Min is %d\n", h_result_min);
  printf("Max is %d\n", h_result_max);


  hipFree(d_A);
  hipFree(d_result_min);
  hipFree(d_result_max);

  return 0;
}
