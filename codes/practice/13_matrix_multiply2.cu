
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 4

__global__ void matrix_multiply(int *da, int *db, int *dc){
  
  int row = blockIdx.x*blockDim.x+threadIdx.x;
  int col = blockIdx.y*blockDim.y+threadIdx.y;


  if (row<N && col<N){
    dc[row*N + col] = 0;
    
    for (int k=0;k<N;k++){
        dc[row*N + col] += da[row*N+k]*db[k*N+col]; 
    }
  }
  
}


int main(){
  
  srand((unsigned int) time(NULL));

  int a[N][N], b[N][N], c[N][N];  
    
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
       a[i][j] = rand()%N;
       b[i][j] = rand()%N;
    }
  }  

  int *da, *db, *dc;  
 
  hipMalloc(&da, N*N*sizeof(int));
  hipMalloc(&db, N*N*sizeof(int));
  hipMalloc(&dc, N*N*sizeof(int));
 

  hipMemcpy(da, a, N*N*sizeof(int), hipMemcpyHostToDevice);   
  hipMemcpy(db, b, N*N*sizeof(int), hipMemcpyHostToDevice);   

  
  int threadsperblockx = N;
  int threadsperblocky = N;

  dim3 threadsperblock(threadsperblockx, threadsperblocky, 1);

  int blockspergridx = (N + threadsperblockx - 1)/threadsperblockx;
  int blockspergridy = (N + threadsperblocky - 1)/threadsperblocky;

  dim3 blockspergrid(blockspergridx, blockspergridy, 1); 

  matrix_multiply<<<threadsperblock,blockspergrid>>>(da, db, dc);
     
  hipMemcpy(c, dc, N*N*sizeof(int), hipMemcpyDeviceToHost);


  printf("\n a: \n");
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
       printf("%d ",a[i][j]);
    }
    printf("\n");
  }  


  printf("\n b: \n");
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
       printf("%d ",b[i][j]);
    }
    printf("\n");
  }  


  printf("\n c: \n");
  for(int i=0;i<N;i++){
    for(int j=0;j<N;j++){
       printf("%d ",c[i][j]);
    }
    printf("\n");
  }  


  hipFree(da);
  hipFree(db);
  hipFree(dc);

  return 0;
}


