
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 48

// 8 blocks, with 6 threads each

__global__ void minmax(int *d_input, int *d_output_min, int *d_output_max){
  
    __shared__ int shareddata[N/6];
  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    if (idx<N)
    {
       shareddata[threadIdx.x] = d_input[idx];
    }
    else
    {
       shareddata[threadIdx.x] = 0;
    }    
  
    __syncthreads();
  
    if (threadIdx.x == 0){
       int min = shareddata[0];
       int max = shareddata[0];
       for (int i=0; i<blockDim.x; i++)
       {
          if (shareddata[i]<min)
               min = shareddata[i]; 
  
          if (shareddata[i]>max)
              max = shareddata[i];  
       }
       d_output_min[blockIdx.x] = min;
       d_output_max[blockIdx.x] = max;
    }

}



int main()
{
    int h_input[N];
    int *d_input, *d_output_min, *d_output_max;
    int h_output_min[N/6], h_output_max[N/6];
  
    for (int i=0; i<N; i++){
       h_input[i] = i+1;
    } 
  
    hipMalloc(&d_input, N*sizeof(int));
    hipMalloc(&d_output_min, N/6*sizeof(int));
    hipMalloc(&d_output_max, N/6*sizeof(int));
  
    hipMemcpy(d_input, h_input, N*sizeof(int), hipMemcpyHostToDevice);
    
    int threadsperblock=6;
    int blockspergrid=(N+threadsperblock-1)/threadsperblock;
    
    minmax<<<blockspergrid, threadsperblock>>>(d_input, d_output_min, d_output_max);
    
    hipDeviceSynchronize();
     
    hipMemcpy(h_output_min, d_output_min, N/6*sizeof(int), hipMemcpyDeviceToHost);  
    hipMemcpy(h_output_max, d_output_max, N/6*sizeof(int), hipMemcpyDeviceToHost);  
  
    int min = h_output_min[0];
    int max = h_output_max[0];
  
    for (int i=0; i<N/6; i++){
      if (h_output_min[i]<min){
          min = h_output_min[i];
      }
  
      if (h_output_max[i]>max){
          max = h_output_max[i];
      }     
    } 
    
    
    printf("Max value is %d \n",max);
    printf("Min value is %d \n",min);
  
    return 0;
}
