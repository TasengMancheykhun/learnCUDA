
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

#define N 3


__global__ void matrix_multiply(int *da, int *db, int *dc){
  
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;     

  if (row < N && col < N){
    dc[row*N+col] = 0;
    for (int k=0; k<N; k++){
       dc[row*N + col] += da[row*N + k]*db[k*N + col];
    }
  }

}



int main(){
 

  int a[N][N], b[N][N], c[N][N]; 
  
  srand((unsigned int )time (NULL));

  for (int i=0; i<N; i++){
      for (int j=0; j<N; j++){
          a[i][j] = rand()%N;  
      }
  }

  for (int i=0; i<N; i++){
      for (int j=0; j<N; j++){
          b[i][j] = rand()%N;  
      }
  }

  int *da, *db, *dc;  

  hipMalloc(&da, N*N*sizeof(int));
  hipMalloc(&db, N*N*sizeof(int));
  hipMalloc(&dc, N*N*sizeof(int));

  hipMemcpy(da, a, N*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(db, b, N*N*sizeof(int), hipMemcpyHostToDevice);

  int threadsperblockx = N;
  int threadsperblocky = N;

  dim3 threadsperblock(threadsperblockx, threadsperblocky, 1);

  int blockspergridx = (N + threadsperblockx - 1)/threadsperblockx;
  int blockspergridy = (N + threadsperblocky - 1)/threadsperblocky;

  dim3 blockspergrid(blockspergridx, blockspergridy, 1);

  matrix_multiply<<<blockspergrid, threadsperblock>>>(da, db, dc); 

  hipMemcpy(c, dc, N*N*sizeof(int), hipMemcpyDeviceToHost);
  
  printf("\na :\n");
  for (int i=0; i<N; i++){
    for (int j=0; j<N; j++){
       printf("%d ",a[i][j]);
    } 
    printf("\n");
  }


  printf("\nb :\n");
  for (int i=0; i<N; i++){
    for (int j=0; j<N; j++){
       printf("%d ",b[i][j]);
    } 
    printf("\n");
  }

  printf("\nc :\n");
  for (int i=0; i<N; i++){
    for (int j=0; j<N; j++){
       printf("%d ",c[i][j]);
    } 
    printf("\n");
  }


  hipFree(da);
  hipFree(db);
  hipFree(dc);

  return 0;
}
