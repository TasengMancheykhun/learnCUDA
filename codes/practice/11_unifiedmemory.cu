#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#define N 32

__global__ void add(int *d_a,int *d_b,int *d_c){
  int idx = blockDim.x*blockIdx.x+threadIdx.x;

  if (idx<N){
    d_c[idx] = d_a[idx] + d_b[idx];
  }


}



int main(){

   int *d_a, *d_b, *d_c;

   hipMallocManaged(&d_a, N*sizeof(int));
   hipMallocManaged(&d_b, N*sizeof(int));
   hipMallocManaged(&d_c, N*sizeof(int));
   
   for(int i=0; i<N; i++){
     d_a[i] = i+1;
     d_b[i] = i+1;
   }
   
   int threadsperblock = 8;
   int blockspergrid = (N+threadsperblock-1)/threadsperblock;
   
   add<<<blockspergrid,threadsperblock>>>(d_a, d_b, d_c);
   hipDeviceSynchronize();

   printf("Result: \n");
   for(int i=0; i<N; i++){
       printf("%d ",d_c[i]);
   }
   printf("\n");

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}

